#include "vram_stats.hpp"

#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    float* pseudo_buffer {nullptr};
    hipMalloc(&pseudo_buffer, 1<<30);
    size_t vram_usage = VramStats::GetTacoUsage();
    printf("VRAM usage %zu bytes\n", vram_usage);
    hipFree(pseudo_buffer);
    return 0;
}